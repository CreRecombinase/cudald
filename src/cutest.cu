#include "hip/hip_runtime.h"





#include <thrust/system/cuda/experimental/pinned_allocator.h>
#include <hipblas.h>
#include <hipsparse.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/reduce.h>
#include <thrust/functional.h>
#include <thrust/random.h>
#include <unistd.h>

#include <thrust/sequence.h>
#define BLOCK_SIZE 16

double calc_theta(const int m){
    double nmsum=0;
    for(int i=1;i<2*m;i++){
        nmsum+=1.0/static_cast<double>(i);
    }
    return((1/nmsum)/(2*m+1/nmsum));
}


__global__
void ldshrink(float* S, const float* mapd, const int p, const float m, const float ne, const float cutoff, const double theta)
{
    int k = threadIdx.x + blockIdx.x * blockDim.x;

    int i = p - 2 - std::floor(std::sqrt(-8*k + 4*p*(p-1)-7)/2.0 - 0.5);
    int j = k + i + 1 - p*(p-1)/2 + (p-i)*((p-i)-1)/2;
    if(i<p&&j<p) {
        auto tsi = S[i * p + i];
        auto tsj = S[j * p + j];

        auto shrinkage = std::exp(-(4 * ne * std::abs(mapd[j] - mapd[i]) / 100) / (2 * m));
        shrinkage = shrinkage < cutoff ? 0 : shrinkage;
        auto tS = 1 / std::sqrt(tsi + 0.5 * theta * (1 - 0.5 * theta)) * ((1 - theta) * (1 - theta)) * S[i * p + j] *
                  shrinkage * (1 / std::sqrt(tsj));
        S[j * p + i] = tS;
        S[i * p + j] = tS;
    }

}
__global__
void zero_diagonal(float* S,const int p) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    S[i*p+i]=1;
}
__global__
void row_diagonal(float* S,const int p) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    S[i*p+i]=i;
}

__global__
void idx_check(float* rowm,float *colm,const int p) {
    int k = threadIdx.x + blockIdx.x * blockDim.x;

    int i = p - 2 - std::floor(std::sqrt(-8*k + 4*p*(p-1)-7)/2.0 - 0.5);
    int j = k + i + 1 - p*(p-1)/2 + (p-i)*((p-i)-1)/2;
    if(i<p&&j<p) {
        rowm[i * p + j] = i;
        rowm[j * p + i] = j;

        colm[i * p + j] = j;
        colm[j * p + i] = i;
    }
}

std::pair<std::vector<float>,std::vector<float> >id_check(const size_t p) {

    int blockSize;   // The launch configurator returned block size
    int minGridSize; // The minimum grid size needed to achieve the
    // maximum occupancy for a full device launch
    int gridSize;
    hipOccupancyMaxPotentialBlockSize( &minGridSize, &blockSize,
                                        idx_check, 0, 0);
    // Round up according to array size
    int arrayCount = (p*p-p)/2;
    gridSize = (arrayCount + blockSize - 1) / blockSize;
    thrust::device_vector<float> d_cov1(p*p);
    thrust::device_vector<float> d_cov2(p*p);
    idx_check<<<gridSize, blockSize>>>(thrust::raw_pointer_cast(d_cov1.data()),thrust::raw_pointer_cast(d_cov2.data()),p);
    row_diagonal<<<1,p>>>(thrust::raw_pointer_cast(d_cov1.data()),p);
    row_diagonal<<<1,p>>>(thrust::raw_pointer_cast(d_cov2.data()),p);
    std::vector<float> res_data1(p*p);
    std::vector<float> res_data2(p*p);
    thrust::copy(d_cov1.begin(),d_cov1.end(),res_data1.begin());
    thrust::copy(d_cov2.begin(),d_cov2.end(),res_data2.begin());
    return(std::make_pair(res_data1,res_data2));
}


    std::vector<float> cuda_cov(float*X, const size_t n, const size_t p,const float* mapd,float m,float ne,float cutoff) {
        hipblasHandle_t handle; // CUBLAS context

    float alpha = 1.0f;
    float beta = 0.0f; // bet =1
    int blockSize;   // The launch configurator returned block size
    int minGridSize; // The minimum grid size needed to achieve the
        // maximum occupancy for a full device launch
    int gridSize;
    hipOccupancyMaxPotentialBlockSize( &minGridSize, &blockSize,
                                            ldshrink, 0, 0);
        int arrayCount = (p*p-p)/2;
        gridSize = (arrayCount + blockSize - 1) / blockSize;

        thrust::device_vector<float> d_cov1(p*p);
    thrust::device_vector<float> d_cov2(p*p);
    thrust::device_vector<float> d_covResult(p*p);
    const thrust::device_vector<float> d_map(mapd,mapd+p);
    thrust::device_vector<float> d_wholeMatrix(X,X+n*p);
    thrust::device_vector<float> d_meansVec(p); // rowVec of means of trials
    float *meanVecPtr = thrust::raw_pointer_cast(d_meansVec.data());
    float *device2DMatrixPtr = thrust::raw_pointer_cast(d_wholeMatrix.data());

    thrust::device_vector<float> deviceVector(n, 1.0f);

        hipblasCreate(&handle);
    auto theta = calc_theta(m);
    auto dimensionSize=p;
    alpha = 1.0f / n;
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, dimensionSize, dimensionSize, n, &alpha,
                device2DMatrixPtr, dimensionSize, device2DMatrixPtr, dimensionSize, &beta,
                thrust::raw_pointer_cast(d_cov1.data()), dimensionSize);

        // Mean vector of each column
        alpha = 1.0f;
        hipblasSgemv(handle, HIPBLAS_OP_N, dimensionSize, n, &alpha, device2DMatrixPtr,
                    dimensionSize, thrust::raw_pointer_cast(deviceVector.data()), 1, &beta, meanVecPtr, 1);

        // MeanVec * transpose(MeanVec) / N*N
        alpha = 1.0f / (n*n);
        hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, dimensionSize, dimensionSize, 1, &alpha,
                    meanVecPtr, 1, meanVecPtr, 1, &beta,
                    thrust::raw_pointer_cast(d_cov2.data()), dimensionSize);

        alpha = 1.0f;
        beta = -1.0f;
        //  (X*transpose(X) / N) -  (MeanVec * transpose(MeanVec) / N*N)
        hipblasSgeam(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, dimensionSize, dimensionSize, &alpha,
                    thrust::raw_pointer_cast(d_cov1.data()), dimensionSize, &beta,
                    thrust::raw_pointer_cast(d_cov2.data()),
                    dimensionSize, thrust::raw_pointer_cast(d_covResult.data()), dimensionSize);

        // Go to other class and calculate its covarianceMatrix


        std::vector<float> res_data(p*p);
        ldshrink<<<gridSize, blockSize>>>(thrust::raw_pointer_cast(d_covResult.data()),thrust::raw_pointer_cast(d_map.data()),p,m,ne,cutoff,theta);
        zero_diagonal<<<1,p>>>(thrust::raw_pointer_cast(d_covResult.data()),p);
        thrust::copy(d_covResult.begin(),d_covResult.end(),res_data.begin());

        hipblasDestroy(handle);
    return res_data;
}
